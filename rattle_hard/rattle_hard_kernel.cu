#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <torch/extension.h>

// CUDA kernels for converting between batched and flattened matrices
__global__ void convertFlattenedToBatched(float *d_x_flattened, float **d_x_ptrs, int batchSize, int matrixSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < batchSize) {
        d_x_ptrs[idx] = d_x_flattened + idx * matrixSize;
    }
}

__global__ void convertBatchedToFlattened(float **d_x_ptrs, float *d_x_flattened, int batchSize, int matrixSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < batchSize) {
        float *matrix = d_x_ptrs[idx];
        for (int i = 0; i < matrixSize; i++) {
            d_x_flattened[idx * matrixSize + i] = matrix[i];
        }
    }
}

// elementwise kernels
__global__ void elementwiseInverse(float* x, float* y, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        if (x[idx] != 0.0f) { // Avoid division by zero
            y[idx] = 1.0f / x[idx];
        } else {
            y[idx] = 0.0f; // Or some other safe value
        }
    }
}

__global__ void elementwiseDiv(float* d_x, float* d_y, float* d_result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        if (d_y[idx] != 0.0f) { // Avoid division by zero
            d_result[idx] = d_x[idx] / d_y[idx];
        } else {
            d_result[idx] = 0.0f; // Or some other safe value
        }
    }
}

__global__ void elementwiseMul(float* d_x, float* d_y, float* d_result, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
            d_result[idx] = d_x[idx] * d_y[idx];
        }
}

__global__ void G(float **x_ptrs, float *output, int batchSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batchSize) return;

    float *x = x_ptrs[idx];  // pointer to batch element (3 floats)
    float result = x[0]*x[0] + x[1]*x[1] + x[2]*x[2] - 1.0f;
    output[idx] = result;
}

void checkCublas(hipblasStatus_t status, const std::string& functionName) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error in " << functionName << ": " << status << std::endl;
        exit(1);
    }
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(-1);  // Exit if error occurs
    }
}


// Function to copy and print a single matrix/vector from device memory:
void printDeviceMatrix(const float* d_mat, int rows, int cols) {
    std::vector<float> h_mat(rows * cols);
    // Synchronous copy from device to host
    hipMemcpy(h_mat.data(), d_mat, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            std::cout << h_mat[r * cols + c] << " ";
        }
        std::cout << std::endl;
    }
}


void printWholeBatchR(float** d_R_ptr, int batchSize, int rows, int cols) {
    for (int i = 0; i < batchSize; ++i) {
        // Copy device pointer for ith matrix from device to host
        float* h_single_R_ptr;
        hipMemcpy(&h_single_R_ptr, d_R_ptr + i, sizeof(float*), hipMemcpyDeviceToHost);

        std::cout << "R_ptr[" << i << "]:" << std::endl;
        printDeviceMatrix(h_single_R_ptr, rows, cols);
        std::cout << std::endl;
    }
}

void rattleHard(hipblasHandle_t handle, hipsolverHandle_t cusolver_handle, float *x, float *v, int batchSize, float h) {
    float alpha = 4.0f;
    float beta = 0.0f; 
    float alpha2 = 2.0f;
    float diffval = -1.0f;
    float h_t = 4.0f/(h*h);
    
    // Device pointers
    float *x_ptr_new, *v_ptr_new, *dL_ptr, *diff_ptr, *v12_ptr;
    float **d_I, **diff_ptrs, **dL_ptrs, **v_ptrs_new;
    float *d_I_flat, *R_ptr_flat, *L_ptrs_flat;
    float **L_ptrs;
    
    // Allocate memory for flat arrays
    float *x_ptr = x;  // Use input device pointer
    float *v_ptr = v;  // Use input device pointer
    checkCudaError(hipMalloc(&x_ptr_new, batchSize * 3 * sizeof(float)));
    checkCudaError(hipMalloc(&v_ptr_new, batchSize * 3 * sizeof(float)));

    checkCudaError(hipMalloc(&d_I_flat, batchSize * sizeof(float)));
    checkCudaError(hipMalloc(&R_ptr_flat, batchSize * sizeof(float)));
    
    // Allocate memory for L_ptrs_flat and L_ptrs - FIX: These were missing
    checkCudaError(hipMalloc(&L_ptrs_flat, batchSize * 3 * sizeof(float)));
    checkCudaError(hipMalloc(&L_ptrs, batchSize * sizeof(float*)));
    
    // Allocate memory for pointer arrays
    checkCudaError(hipMalloc(&diff_ptrs, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&dL_ptrs, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&d_I, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&v_ptrs_new, batchSize * sizeof(float*)));

    // Allocate memory for other variables
    checkCudaError(hipMalloc(&dL_ptr, batchSize * 3 * sizeof(float))); // FIX: Size should be 3*batchSize
    checkCudaError(hipMalloc(&diff_ptr, batchSize * 3 * sizeof(float)));
    checkCudaError(hipMalloc(&v12_ptr, batchSize * 3 * sizeof(float)));
    
    // Copy data to device
    checkCudaError(hipMemcpy(x_ptr, x, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(v_ptr, v, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(x_ptr_new, x, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice));
    
    // Add h*v to x_new
    checkCublas(hipblasSaxpy(handle, batchSize * 3, &h, v_ptr, 1, x_ptr_new, 1), "hipblasSaxpy 1");
    
    // Allocate memory for the batched pointers
    float **x_ptrs, **x_ptrs_new, **R_ptr;
    checkCudaError(hipMalloc(&x_ptrs, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&x_ptrs_new, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&R_ptr, batchSize * sizeof(float*)));
    
    // Set up host pointers
    float** x_ptrs_host = new float*[batchSize];
    float** x_ptrs_new_host = new float*[batchSize];
    float** R_ptr_host = new float*[batchSize];
    float** diff_ptrs_host = new float*[batchSize];
    float** dL_ptrs_host = new float*[batchSize];
    float** d_I_host = new float*[batchSize];
    float** L_ptrs_host = new float*[batchSize];  // FIX: Added for L_ptrs
    float** v_ptrs_new_host = new float*[batchSize]; // FIX: Added for v_ptrs_new
    
    // Allocate memory for each matrix
    for (int i = 0; i < batchSize; i++) {
        checkCudaError(hipMalloc(&x_ptrs_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&x_ptrs_new_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&R_ptr_host[i], sizeof(float)));
        checkCudaError(hipMalloc(&diff_ptrs_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&dL_ptrs_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&d_I_host[i], sizeof(float)));
        checkCudaError(hipMalloc(&L_ptrs_host[i], 3 * sizeof(float))); // FIX: Added for L_ptrs
        checkCudaError(hipMalloc(&v_ptrs_new_host[i], 3 * sizeof(float))); // FIX: Added for v_ptrs_new
        
        // Initialize d_I with value 1.0f
        float one = 1.0f;
        checkCudaError(hipMemcpy(d_I_host[i], &one, sizeof(float), hipMemcpyHostToDevice));
        
        // Copy individual matrix data to device
        checkCudaError(hipMemcpy(x_ptrs_host[i], x + i * 3, 3 * sizeof(float), hipMemcpyHostToDevice));
        checkCudaError(hipMemcpy(x_ptrs_new_host[i], x_ptr_new + i * 3, 3 * sizeof(float), hipMemcpyHostToDevice));
    }
    
    // Copy pointer arrays to device
    checkCudaError(hipMemcpy(x_ptrs, x_ptrs_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(x_ptrs_new, x_ptrs_new_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(R_ptr, R_ptr_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(diff_ptrs, diff_ptrs_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(dL_ptrs, dL_ptrs_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_I, d_I_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(L_ptrs, L_ptrs_host, batchSize * sizeof(float*), hipMemcpyHostToDevice)); // FIX: Added for L_ptrs
    checkCudaError(hipMemcpy(v_ptrs_new, v_ptrs_new_host, batchSize * sizeof(float*), hipMemcpyHostToDevice)); // FIX: Added for v_ptrs_new
        
    // Set up kernel execution parameters
    int blockSize = 256;
    int numBlocks = (batchSize + blockSize - 1) / blockSize;
    float* diff_host = new float[batchSize * 3];
    float *output_flat;
    checkCudaError(hipMalloc(&output_flat, batchSize * sizeof(float)));
    // Main calculation loop
    for (int i = 0; i < 3; i++) {
        // First matrix multiplication
        // FIX: Correct matrix dimensions - assuming each x is a 1x3 row vector
        checkCublas(hipblasSgemmBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_T, // FIX: Transpose second matrix
            1, 1, 3,
            &alpha, // multiplying result by 4 for the sphere
            (const float**)x_ptrs, 1,
            (const float**)x_ptrs_new, 1, // FIX: Leading dimension should be 1 for row vector
            &beta,
            R_ptr, 1,
            batchSize), "hipblasSgemmBatched 1");
        
        checkCudaError(hipDeviceSynchronize());
        // Convert batched to flattened
        convertBatchedToFlattened<<<numBlocks, blockSize>>>(R_ptr, R_ptr_flat, batchSize, 1);
        checkCudaError(hipDeviceSynchronize());
        
        // Compute elementwise inverse
        elementwiseInverse<<<numBlocks, blockSize>>>(R_ptr_flat, d_I_flat, batchSize);
        checkCudaError(hipDeviceSynchronize());
        // printDeviceMatrix(d_I_flat, 1, 1);  // if your matrices are 1x1 as in your example

        // Convert flattened to batched
        convertFlattenedToBatched<<<numBlocks, blockSize>>>(d_I_flat, d_I, batchSize, 1);
        checkCudaError(hipDeviceSynchronize());
        
        G<<<numBlocks, blockSize>>>(x_ptrs_new, output_flat, batchSize);
        elementwiseMul<<<numBlocks, blockSize>>>(d_I_flat, output_flat, dL_ptr, batchSize);
        convertFlattenedToBatched<<<numBlocks, blockSize>>>(dL_ptr, dL_ptrs, batchSize, 1);

        
        checkCudaError(hipDeviceSynchronize());
        
        // Third matrix multiplication
        checkCublas(hipblasSgemmBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            3, 1, 1,
            &alpha2,
            (const float**)x_ptrs, 3,
            (const float**)dL_ptrs, 1,
            &beta,
            diff_ptrs, 3,
            batchSize), "hipblasSgemmBatched 3"); // diff = 2 x dL_ptrs


        checkCudaError(hipDeviceSynchronize());
        
        // Convert batched to flattened for diff
        convertBatchedToFlattened<<<numBlocks, blockSize>>>(diff_ptrs, diff_ptr, batchSize, 3);
        checkCudaError(hipDeviceSynchronize());

        
        // checkCudaError(hipMemcpy(diff_host, diff_ptr, batchSize * 3 * sizeof(float), hipMemcpyDeviceToHost));

        // // Print diff_host contents
        // std::cout << "diff_ptr after iteration " << i << ":\n";
        // for (int b = 0; b < batchSize; b++) {
        //     std::cout << "Batch " << b << ": ";
        //     for (int j = 0; j < 3; j++) {
        //         std::cout << diff_host[b * 3 + j] << " ";
        //     }
        //     std::cout << "\n";
        // }
        // Apply diff to x_ptr_new
        checkCublas(hipblasSaxpy(handle, batchSize * 3, &diffval, diff_ptr, 1, x_ptr_new, 1), "hipblasSaxpy 2"); //xnew​=xnew​- diff_ptr
        convertFlattenedToBatched<<<numBlocks, blockSize>>>(x_ptr_new, x_ptrs_new, batchSize, 1);

    }
    
    // Copy x_new to v_new
    checkCudaError(hipMemcpy(v_ptr_new, x_ptr_new, 3 * batchSize * sizeof(float), hipMemcpyDeviceToDevice));
    
    // Subtract original x from v_new
    checkCublas(hipblasSaxpy(handle, batchSize * 3, &diffval, x_ptr, 1, v_ptr_new, 1), "hipblasSaxpy 3"); // vptr_new = xptr_new - x_ptr . still needs to be divided by h
    
    // float* v_host = new float[batchSize * 3];
    // hipMemcpy(v_host, v_ptr_new, batchSize * 3 * sizeof(float), hipMemcpyDeviceToHost);


    // Convert flattened to batched for updated x
    convertFlattenedToBatched<<<numBlocks, blockSize>>>(x_ptr_new, x_ptrs_new, batchSize, 3);
    checkCudaError(hipDeviceSynchronize());
    
    // Fourth matrix multiplication
    // FIX: Correct dot product dimensions
    checkCublas(hipblasSgemmBatched(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_T, // FIX: Transpose second matrix
        1, 1, 3,
        &alpha,
        (const float**)x_ptrs_new, 1,
        (const float**)x_ptrs_new, 1,
        &beta,
        R_ptr, 1,
        batchSize), "hipblasSgemmBatched 4"); // updating R_ptr, this is equivalent to P in the original
    
    checkCudaError(hipDeviceSynchronize());
    
    // Convert flattened to batched for v_new
    convertFlattenedToBatched<<<numBlocks, blockSize>>>(v_ptr_new, v_ptrs_new, batchSize, 3);
    checkCudaError(hipDeviceSynchronize());
    
    // Fifth matrix multiplication
    checkCublas(hipblasSgemmBatched(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        3, 1, 1,
        &h_t,
        (const float**)x_ptrs_new, 3,
        (const float**)v_ptrs_new, 1,
        &beta,
        dL_ptrs, 3,
        batchSize), "hipblasSgemmBatched 5"); // dl_ptrs = 4(x @ 2v/h) - equal to t in the original implementation
        // h_t is 4/h^2 because v still needed to be divided by h, 
    
    checkCudaError(hipDeviceSynchronize());
    
    // Convert batched to flattened
    convertBatchedToFlattened<<<numBlocks, blockSize>>>(R_ptr, R_ptr_flat, batchSize, 1);
    convertBatchedToFlattened<<<numBlocks, blockSize>>>(dL_ptrs, dL_ptr, batchSize, 3);
    checkCudaError(hipDeviceSynchronize());
    
    // Element-wise division
    elementwiseDiv<<<numBlocks, blockSize>>>(R_ptr_flat, dL_ptr, L_ptrs_flat, batchSize * 3); // L = L_ptrs/R = T/P
    checkCudaError(hipDeviceSynchronize());
    
    // Convert flattened to batched
    convertFlattenedToBatched<<<numBlocks, blockSize>>>(L_ptrs_flat, L_ptrs, batchSize, 3);
    checkCudaError(hipDeviceSynchronize());
    
    float h2 = h;
    float h3 = 1.0f/h;
    
    // Final matrix multiplication
    checkCublas(hipblasSgemmBatched(handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        3, 1, 1,
        &h2,
        (const float**)x_ptrs_new, 3,
        (const float**)L_ptrs, 1,
        &h3,
        v_ptrs_new, 3,
        batchSize), "hipblasSgemmBatched 6"); // v_ptrs_new = (h) * x_ptrs_new * dL_ptrs + 1/h v_ptrs_new = v + h/2 J @ L 
    
    checkCudaError(hipDeviceSynchronize());
    hipMemcpy(x_ptr, x_ptr_new, batchSize * 3 * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(v_ptr, v_ptr_new, batchSize * 3 * sizeof(float), hipMemcpyDeviceToDevice);

    // Free all allocated memory
    // checkCudaError(hipFree(x_ptr));
    // checkCudaError(hipFree(v_ptr));
    checkCudaError(hipFree(x_ptr_new));
    checkCudaError(hipFree(v_ptr_new));
    checkCudaError(hipFree(d_I_flat));
    checkCudaError(hipFree(R_ptr_flat));
    checkCudaError(hipFree(L_ptrs_flat));  // FIX: Free allocated memory
    checkCudaError(hipFree(diff_ptrs));
    checkCudaError(hipFree(dL_ptrs));
    checkCudaError(hipFree(d_I));
    checkCudaError(hipFree(L_ptrs));       // FIX: Free allocated memory
    checkCudaError(hipFree(v_ptrs_new));   // FIX: Free allocated memory
    checkCudaError(hipFree(dL_ptr));
    checkCudaError(hipFree(diff_ptr));
    checkCudaError(hipFree(v12_ptr));
    checkCudaError(hipFree(x_ptrs));
    checkCudaError(hipFree(x_ptrs_new));
    checkCudaError(hipFree(R_ptr));
    
    // Free host temporary arrays
    for (int i = 0; i < batchSize; i++) {
        checkCudaError(hipFree(x_ptrs_host[i]));
        checkCudaError(hipFree(x_ptrs_new_host[i]));
        checkCudaError(hipFree(R_ptr_host[i]));
        checkCudaError(hipFree(diff_ptrs_host[i]));
        checkCudaError(hipFree(dL_ptrs_host[i]));
        checkCudaError(hipFree(d_I_host[i]));
        checkCudaError(hipFree(L_ptrs_host[i]));     // FIX: Free allocated memory
        checkCudaError(hipFree(v_ptrs_new_host[i])); // FIX: Free allocated memory
    }
    
    delete[] x_ptrs_host;
    delete[] x_ptrs_new_host;
    delete[] R_ptr_host;
    delete[] diff_ptrs_host;
    delete[] dL_ptrs_host;
    delete[] d_I_host;
    delete[] L_ptrs_host;     // FIX: Delete allocated memory
    delete[] v_ptrs_new_host; // FIX: Delete allocated memory
}


void rattle_hard_launcher(torch::Tensor x, torch::Tensor v, float h) {
    TORCH_CHECK(x.is_cuda(), "x must be a CUDA tensor");
    TORCH_CHECK(v.is_cuda(), "v must be a CUDA tensor");
    TORCH_CHECK(x.dtype() == torch::kFloat32, "x must be float32");
    TORCH_CHECK(v.dtype() == torch::kFloat32, "v must be float32");

    float* d_x = x.data_ptr<float>();
    float* d_v = v.data_ptr<float>();
    int batchSize = x.size(0);

    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    hipsolverHandle_t cusolverHandle;
    hipsolverDnCreate(&cusolverHandle);

    // Pass device pointers directly to rattleHard
    rattleHard(cublasHandle, cusolverHandle, d_x, d_v, batchSize, h);

    hipblasDestroy(cublasHandle);
    hipsolverDnDestroy(cusolverHandle);
}

int main() {
    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    std::cout << "maxThreadsPerBlock: " << maxThreadsPerBlock << std::endl;
    
    // Initialize cuBLAS and cuSolver handles
    hipblasHandle_t cublasHandle;
    hipsolverHandle_t cusolverHandle;
    
    // Create handles
    hipblasCreate(&cublasHandle);
    hipsolverDnCreate(&cusolverHandle);
    
    // Define batch size and the value for h
    int batchSize = 2; // Example batch size
    float h = 0.01f;   // Example step size
    
    // Example x and v data (batchSize x 3, 3D matrix per batch)
    float h_x[batchSize * 3] = {1.0f, 2.0f, 3.0f,  // x[0]
                                4.0f, 5.0f, 6.0f}; // x[1]
    float h_v[batchSize * 3] = {0.5f, 0.5f, 0.5f,  // v[0]
                                0.1f, 0.1f, 0.1f}; // v[1]
    
    // Allocate device memory for x and v
    float *d_x, *d_v;
    hipMalloc(&d_x, batchSize * 3 * sizeof(float));
    hipMalloc(&d_v, batchSize * 3 * sizeof(float));
    
    // Copy data from host to device
    hipMemcpy(d_x, h_x, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice);
    
    // Call the rattleHard function
    rattleHard(cublasHandle, cusolverHandle, d_x, d_v, batchSize, h);
    
    // Copy the result back to host
    hipMemcpy(h_x, d_x, batchSize * 3 * sizeof(float), hipMemcpyDeviceToHost);
    

    // Cleanup
    hipFree(d_x);
    hipFree(d_v);
    hipblasDestroy(cublasHandle);
    hipsolverDnDestroy(cusolverHandle);
    
    return 0;
}