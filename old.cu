
void batchedMultiply(hipblasHandle_t handle, float *d_A, float *d_B, float *d_C, int batchSize, int m, int n, int k) {
    float alpha = 1.0f, beta = 0.0f;

    float *d_A_array[] = { d_A, d_A + m * k };
    float *d_B_array[] = { d_B, d_B + k * n };
    float *d_C_array[] = { d_C, d_C + m * n };

    float **d_A_ptr, **d_B_ptr, **d_C_ptr;
    hipMalloc(&d_A_ptr, batchSize * sizeof(float *)); // why are the pointers passed like this?
    hipMalloc(&d_B_ptr, batchSize * sizeof(float *));
    hipMalloc(&d_C_ptr, batchSize * sizeof(float *));

    hipMemcpy(d_A_ptr, d_A_array, batchSize * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(d_B_ptr, d_B_array, batchSize * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(d_C_ptr, d_C_array, batchSize * sizeof(float *), hipMemcpyHostToDevice);

    checkCublas(hipblasSgemmBatched(handle,
                                   HIPBLAS_OP_N, HIPBLAS_OP_N, // specifying no transpose
                                   m, n, k,
                                   &alpha,
                                   d_A_ptr, m,
                                   d_B_ptr, k,
                                   &beta,
                                   d_C_ptr, m,
                                   batchSize));

    hipFree(d_A_ptr);
    hipFree(d_B_ptr);
    hipFree(d_C_ptr);  // why is d_C_ptr is freed, how can I get the result? shouldn't it be written to host first?
}


void batchedMultiplyTranspose(hipblasHandle_t handle, float *d_A, float *d_B, float *d_C, int batchSize, int m, int n, int k) {
    float alpha = 1.0f, beta = 0.0f;

    float *d_A_array[] = { d_A, d_A + m * k };
    float *d_B_array[] = { d_B, d_B + n * k }; // this will be transposed before the multiplication
    float *d_C_array[] = { d_C, d_C + m * n };

    float **d_A_ptr, **d_B_ptr, **d_C_ptr;
    hipMalloc(&d_A_ptr, batchSize * sizeof(float *)); // why are the pointers passed like this?
    hipMalloc(&d_B_ptr, batchSize * sizeof(float *));
    hipMalloc(&d_C_ptr, batchSize * sizeof(float *));

    hipMemcpy(d_A_ptr, d_A_array, batchSize * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(d_B_ptr, d_B_array, batchSize * sizeof(float *), hipMemcpyHostToDevice);
    hipMemcpy(d_C_ptr, d_C_array, batchSize * sizeof(float *), hipMemcpyHostToDevice);

    checkCublas(hipblasSgemmBatched(handle,
                                   HIPBLAS_OP_N, HIPBLAS_OP_T, // specifying no transpose
                                   m, n, k,
                                   &alpha,
                                   d_A_ptr, m,
                                   d_B_ptr, k,
                                   &beta,
                                   d_C_ptr, m,
                                   batchSize));

    hipFree(d_A_ptr);
    hipFree(d_B_ptr);
    hipFree(d_C_ptr);  // why is d_C_ptr is freed, how can I get the result? shouldn't it be written to host first?
}
