#include "hip/hip_runtime.h"
#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

// CUDA kernels for converting between batched and flattened matrices
__global__ void convertFlattenedToBatched(float *d_x_flattened, float **d_x_ptrs, int batchSize, int matrixSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < batchSize) {
        d_x_ptrs[idx] = d_x_flattened + idx * matrixSize;
    }
}

__global__ void convertBatchedToFlattened(float **d_x_ptrs, float *d_x_flattened, int batchSize, int matrixSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < batchSize) {
        float *matrix = d_x_ptrs[idx];
        for (int i = 0; i < matrixSize; i++) {
            d_x_flattened[idx * matrixSize + i] = matrix[i];
        }
    }
}

__global__ void elementwise_inverse(float* x, float* y, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        if (x[idx] != 0.0f) { // Avoid division by zero
            y[idx] = 1.0f / x[idx];
        } else {
            y[idx] = 0.0f; // Or some other safe value
        }
    }
}

void checkCublas(hipblasStatus_t status, const std::string& functionName) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS Error in " << functionName << "!" << std::endl;
        exit(1);
    }
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(-1);  // Exit if error occurs
    }
}

void rattleHard(hipblasHandle_t handle, hipsolverHandle_t cusolver_handle, float *x, float *v, int batchSize, float h) {
    float alpha = 4.0f;
    float beta = 0.0f; 
    float alpha2 = 2.0f;
    float diffval = -1.0f;
    
    // Device pointers
    float *x_ptr, *v_ptr, *x_ptr_new, *dL_ptr, *diff_ptr, *v12_ptr;
    float **d_I, **diff_ptrs, **dL_ptrs;
    float *d_I_flat, *R_ptr_flat;
    
    // Allocate memory for flat arrays
    checkCudaError(hipMalloc(&x_ptr, batchSize * 3 * sizeof(float)));
    checkCudaError(hipMalloc(&v_ptr, batchSize * 3 * sizeof(float)));
    checkCudaError(hipMalloc(&x_ptr_new, batchSize * 3 * sizeof(float)));
    checkCudaError(hipMalloc(&d_I_flat, batchSize * sizeof(float)));
    checkCudaError(hipMalloc(&R_ptr_flat, batchSize * sizeof(float)));
    
    // Allocate memory for pointer arrays
    checkCudaError(hipMalloc(&diff_ptrs, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&dL_ptrs, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&d_I, batchSize * sizeof(float*)));
    
    // Allocate memory for other variables
    checkCudaError(hipMalloc(&dL_ptr, batchSize * sizeof(float)));
    checkCudaError(hipMalloc(&diff_ptr, batchSize * 3 * sizeof(float))); // Changed size to 3*batchSize
    checkCudaError(hipMalloc(&v12_ptr, batchSize * 3 * sizeof(float))); // Changed size to 3*batchSize
    
    // Copy data to device
    checkCudaError(hipMemcpy(x_ptr, x, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(v_ptr, v, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(x_ptr_new, x, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice));
    
    // Add h*v to x_new
    checkCublas(hipblasSaxpy(handle, batchSize * 3, &h, v_ptr, 1, x_ptr_new, 1), "hipblasSaxpy 1");
    
    // Allocate memory for the batched pointers
    float **x_ptrs, **x_ptrs_new, **R_ptr;
    checkCudaError(hipMalloc(&x_ptrs, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&x_ptrs_new, batchSize * sizeof(float*)));
    checkCudaError(hipMalloc(&R_ptr, batchSize * sizeof(float*)));
    
    // Set up host pointers and individual matrices
    float** x_ptrs_host = new float*[batchSize];
    float** x_ptrs_new_host = new float*[batchSize];
    float** R_ptr_host = new float*[batchSize];
    float** diff_ptrs_host = new float*[batchSize];
    float** dL_ptrs_host = new float*[batchSize];
    float** d_I_host = new float*[batchSize];
    
    // Allocate memory for each matrix
    for (int i = 0; i < batchSize; i++) {
        checkCudaError(hipMalloc(&x_ptrs_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&x_ptrs_new_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&R_ptr_host[i], sizeof(float)));
        checkCudaError(hipMalloc(&diff_ptrs_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&dL_ptrs_host[i], 3 * sizeof(float)));
        checkCudaError(hipMalloc(&d_I_host[i], sizeof(float)));
        
        // Initialize d_I with value 1.0f
        float one = 1.0f;
        checkCudaError(hipMemcpy(d_I_host[i], &one, sizeof(float), hipMemcpyHostToDevice));
        
        // Copy individual matrix data to device
        checkCudaError(hipMemcpy(x_ptrs_host[i], x + i * 3, 3 * sizeof(float), hipMemcpyHostToDevice));
        checkCudaError(hipMemcpy(x_ptrs_new_host[i], x_ptr_new + i * 3, 3 * sizeof(float), hipMemcpyHostToDevice));
    }
    
    // Copy pointer arrays to device
    checkCudaError(hipMemcpy(x_ptrs, x_ptrs_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(x_ptrs_new, x_ptrs_new_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(R_ptr, R_ptr_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(diff_ptrs, diff_ptrs_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(dL_ptrs, dL_ptrs_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_I, d_I_host, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    
    std::cout << "All memory allocated and initialized successfully." << std::endl;
    
    // Set up kernel execution parameters
    int blockSize = 256;
    int numBlocks = (batchSize + blockSize - 1) / blockSize;
    
    // Main calculation loop - was previously missing proper bracket structure
    for (int i = 0; i < 1; i++) { // Reduced to 1 iteration for simplicity
        // First matrix multiplication
        checkCublas(hipblasSgemmBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            1, 1, 3,
            &alpha,
            x_ptrs, 1,
            x_ptrs_new, 3,
            &beta,
            R_ptr, 3,
            batchSize), "hipblasSgemmBatched 1");
        
        checkCudaError(hipDeviceSynchronize());
        
        // Convert batched to flattened
        convertBatchedToFlattened<<<numBlocks, blockSize>>>(R_ptr, R_ptr_flat, batchSize, 1);
        checkCudaError(hipDeviceSynchronize());
        
        // Compute elementwise inverse
        elementwise_inverse<<<numBlocks, blockSize>>>(R_ptr_flat, d_I_flat, batchSize);
        checkCudaError(hipDeviceSynchronize());
        
        // Convert flattened to batched
        convertFlattenedToBatched<<<numBlocks, blockSize>>>(d_I_flat, d_I, batchSize, 1);
        checkCudaError(hipDeviceSynchronize());
        
        // Second matrix multiplication
        checkCublas(hipblasSgemmBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            1, 3, 1,
            &alpha2,
            d_I, 1,
            x_ptrs_new,3,
            &beta,
            dL_ptrs, 3,
            batchSize), "hipblasSgemmBatched 2");
        
        checkCudaError(hipDeviceSynchronize());
        
        // Third matrix multiplication
        checkCublas(hipblasSgemmBatched(handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            3, 1, 1,
            &alpha2,
            x_ptrs, 3,
            dL_ptrs, 1,
            &beta,
            diff_ptrs, 3,
            batchSize), "hipblasSgemmBatched 3");
        
        checkCudaError(hipDeviceSynchronize());
        
        // Convert batched to flattened for diff
        convertBatchedToFlattened<<<numBlocks, blockSize>>>(diff_ptrs, diff_ptr, batchSize, 3);
        checkCudaError(hipDeviceSynchronize());
        
        // Apply diff to x_ptr_new
        checkCublas(hipblasSaxpy(handle, batchSize * 3, &diffval, diff_ptr, 1, x_ptr_new, 1), "hipblasSaxpy 2");
        
        std::cout << "Finished iteration " << i + 1 << "." << std::endl;
    }
    
    // Update x with v12_ptr (properly initialized)
    for (int i = 0; i < batchSize * 3; i++) {
        float h_v12 = 0.0f; // Initialize with a safe value
        checkCudaError(hipMemcpy(v12_ptr + i, &h_v12, sizeof(float), hipMemcpyHostToDevice));
    }
    
    // Copy result back
    checkCudaError(hipMemcpy(x, x_ptr_new, batchSize * 3 * sizeof(float), hipMemcpyDeviceToHost));
    
    // Free all allocated memory
    checkCudaError(hipFree(x_ptr));
    checkCudaError(hipFree(v_ptr));
    checkCudaError(hipFree(x_ptr_new));
    checkCudaError(hipFree(d_I_flat));
    checkCudaError(hipFree(R_ptr_flat));
    checkCudaError(hipFree(diff_ptrs));
    checkCudaError(hipFree(dL_ptrs));
    checkCudaError(hipFree(d_I));
    checkCudaError(hipFree(dL_ptr));
    checkCudaError(hipFree(diff_ptr));
    checkCudaError(hipFree(v12_ptr));
    checkCudaError(hipFree(x_ptrs));
    checkCudaError(hipFree(x_ptrs_new));
    checkCudaError(hipFree(R_ptr));
    
    // Free host temporary arrays
    for (int i = 0; i < batchSize; i++) {
        checkCudaError(hipFree(x_ptrs_host[i]));
        checkCudaError(hipFree(x_ptrs_new_host[i]));
        checkCudaError(hipFree(R_ptr_host[i]));
        checkCudaError(hipFree(diff_ptrs_host[i]));
        checkCudaError(hipFree(dL_ptrs_host[i]));
        checkCudaError(hipFree(d_I_host[i]));
    }
    
    delete[] x_ptrs_host;
    delete[] x_ptrs_new_host;
    delete[] R_ptr_host;
    delete[] diff_ptrs_host;
    delete[] dL_ptrs_host;
    delete[] d_I_host;
}

int main() {
    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    std::cout << "maxThreadsPerBlock: " << maxThreadsPerBlock << std::endl;
    
    // Initialize cuBLAS and cuSolver handles
    hipblasHandle_t cublasHandle;
    hipsolverHandle_t cusolverHandle;
    
    // Create handles
    hipblasCreate(&cublasHandle);
    hipsolverDnCreate(&cusolverHandle);
    
    // Define batch size and the value for h
    int batchSize = 2; // Example batch size
    float h = 0.01f;   // Example step size
    
    // Example x and v data (batchSize x 3, 3D matrix per batch)
    float h_x[batchSize * 3] = {1.0f, 2.0f, 3.0f,  // x[0]
                                4.0f, 5.0f, 6.0f}; // x[1]
    float h_v[batchSize * 3] = {0.5f, 0.5f, 0.5f,  // v[0]
                                0.1f, 0.1f, 0.1f}; // v[1]
    
    // Allocate device memory for x and v
    float *d_x, *d_v;
    hipMalloc(&d_x, batchSize * 3 * sizeof(float));
    hipMalloc(&d_v, batchSize * 3 * sizeof(float));
    
    // Copy data from host to device
    hipMemcpy(d_x, h_x, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, batchSize * 3 * sizeof(float), hipMemcpyHostToDevice);
    
    // Call the rattleHard function
    rattleHard(cublasHandle, cusolverHandle, d_x, d_v, batchSize, h);
    
    // Copy the result back to host
    hipMemcpy(h_x, d_x, batchSize * 3 * sizeof(float), hipMemcpyDeviceToHost);
    
    // Output the result
    std::cout << "Updated x:" << std::endl;
    for (int i = 0; i < batchSize; ++i) {
        std::cout << "x[" << i << "]: ";
        for (int j = 0; j < 3; ++j) {
            std::cout << h_x[i * 3 + j] << " ";
        }
        std::cout << std::endl;
    }
    
    // Cleanup
    hipFree(d_x);
    hipFree(d_v);
    hipblasDestroy(cublasHandle);
    hipsolverDnDestroy(cusolverHandle);
    
    return 0;
}